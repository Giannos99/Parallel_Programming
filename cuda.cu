#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "./GoL.h"
#include "lcutil.h"
#include "timestamp.h"

__global__ void CycleRoutineGPU(char *CurrentState , char *NextState , int X , int Dimension){
	
	int sum=0;
	int id=blockIdx.x*blockDim.x + threadIdx.x;
	
	if (id<Dimension) {
		
		if(id+X<Dimension ) {
			sum += CurrentState[id+X];
		}
		if(id-X>=0){
			sum += CurrentState[id-X];
		}
		if(id/X == (id+1)/X) {
			sum += CurrentState[id+1];
		}
		if(id/X == (id-1)/X) {
			sum += CurrentState[id-1];
		}
		if(id+X<Dimension && (id+X)/X == (id+X+1)/X) {
			sum += CurrentState[id+X+1];
		}
		if(id+X<Dimension && (id+X)/X == (id+X-1)/X) {
			sum += CurrentState[id+X-1];
		}
		if(id-X>=0 && (id-X)/X == (id-X+1)/X) {
			sum += CurrentState[id-X+1];
		}
		if(id-X>=0 && (id-X)/X == (id-X-1)/X) {
			sum += CurrentState[id-X-1];
		}
		

		if (sum < 2 || sum > 3)
			NextState[id] = 0;
		else if (sum == 3)
			NextState[id] =  1;
		else
			NextState[id] = CurrentState[id];
					
	}
	
	__syncthreads();
}



void CycleRoutineCPU(char *CurrentState , char *NextState , int X , int Dimension){
	
	int sum=0;
	int id;
	for(id = 0 ; id < Dimension ; id++){
		if (id<Dimension) {
		
			if(id+X<Dimension ) {
				sum += CurrentState[id+X];
			}
			if(id-X>=0){
				sum += CurrentState[id-X];
			}
			if(id/X == (id+1)/X) {
				sum += CurrentState[id+1];
			}
			if(id/X == (id-1)/X) {
				sum += CurrentState[id-1];
			}
			if(id+X<Dimension && (id+X)/X == (id+X+1)/X) {
				sum += CurrentState[id+X+1];
			}
			if(id+X<Dimension && (id+X)/X == (id+X-1)/X) {
				sum += CurrentState[id+X-1];
			}
			if(id-X>=0 && (id-X)/X == (id-X+1)/X) {
				sum += CurrentState[id-X+1];
			}
			if(id-X>=0 && (id-X)/X == (id-X-1)/X) {
				sum += CurrentState[id-X-1];
			}
		

			if (sum < 2 || sum > 3)
				NextState[id] = 0;
			else if (sum == 3)
				NextState[id] =  1;
			else
				NextState[id] = CurrentState[id];
					
		}
	}
}


void CycleGPU(char *grid,int X , int Generations ){
	char *CurrentState,*NextState;
	int bytes=X*X*sizeof(char);
	int i;
	
	hipMalloc((void**)&CurrentState,bytes);
	hipMalloc((void**)&NextState,bytes);
	for(i=0;i<Generations;i++){
		hipMemcpy( CurrentState , grid , bytes , hipMemcpyHostToDevice );
		dim3 NumberOfThreads(X);
		dim3 NumberOfBlocks(X);
		CycleRoutineGPU<<<NumberOfBlocks,NumberOfThreads>>>(CurrentState,NextState,X,X*X);
		hipMemcpy(grid,NextState,bytes,hipMemcpyDeviceToHost);
	}
	hipFree(NextState);
	hipFree(CurrentState);
}

void CycleCPU(char *grid,int X , int Generations ){
	char *NextState;
	int bytes=X*X*sizeof(char);
	int i;
	
	//CurrentState = (char*)malloc(sizeof(char*)*X*X);
	NextState = (char*)malloc(sizeof(char*)*X*X);
	
	
	
	for(i=0;i<Generations;i++){
		CycleRoutineCPU(grid,NextState,X,X*X);
		memcpy( grid , NextState , bytes );
	}
}

int main(int argc , char **argv){
	int X,Generations,i,z;
	double cpu,gpu;
	timestamp t_start;
	X=atoi(argv[1]);
	Generations=atoi(argv[2]);
	
	char *grid_Gpu;
	char *grid_Cpu;

	grid_Gpu=(char*)malloc(sizeof(char*)*X*X);
	grid_Cpu=(char*)malloc(sizeof(char*)*X*X);
	
	srand(time(NULL));
	
	for(i=0;i<X;i++){
		for(z=0;z<X;z++){
			grid_Cpu[ i*X + z ] = grid_Gpu[ i*X + z ] = rand()%2;
		}
	}
	t_start = getTimestamp();
	CycleGPU(grid_Gpu , X, Generations );
	gpu = getElapsedtime(t_start);
	t_start = getTimestamp();
	CycleCPU(grid_Cpu , X, Generations );
	cpu = getElapsedtime(t_start);	
	free( grid_Gpu );
	free( grid_Cpu );
	printf("parallel: gens=%d dim=%d   ms %f   \n",Generations, X, gpu );
	printf("serial: gens=%d dim=%d   ms %f   \n\n",Generations, X, cpu );
	return 0;
}
